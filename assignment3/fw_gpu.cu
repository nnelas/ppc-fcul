#include "hip/hip_runtime.h"
#include <assert.h>
#include <bits/c++config.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "workshop.h"

#define GRAPH_SIZE 2000

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

#define THREADS_PER_BLOCK 256
#define BLOCKS MIN(32, (GRAPH_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK)

void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40;
        if (r > 20) {
          r = INF;
        }

        D(i, j) = r;
      }
    }
  }
}

__global__ void floyd_gpu_compute(const int *graph, int graph_size, int *output) {
  int i, j, k, index, stride;
  for (k = 0; k < graph_size; k++) {
    index = blockIdx.x * blockDim.x + threadIdx.x;
    stride = blockDim.x * gridDim.x;
    for (i = index; i < graph_size; i += stride) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}


void floyd_warshall_gpu(const int *graph, int graph_size, int *output) {
  int *dev_output;
  int size;

  size = sizeof(int) * graph_size * graph_size;
  HANDLE_ERROR( hipMalloc(&dev_output, size) );

  hipMemcpy(dev_output, graph, size, hipMemcpyHostToDevice);

  floyd_gpu_compute<<<BLOCKS, THREADS_PER_BLOCK>>>(graph, graph_size, dev_output);

  hipMemcpy(output, dev_output, size, hipMemcpyDeviceToHost);
  hipFree(dev_output);
}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}

int main(int argc, char **argv) {
#define TIMER_START() gettimeofday(&tv1, NULL)
#define TIMER_STOP()                                                           \
  gettimeofday(&tv2, NULL);                                                    \
  timersub(&tv2, &tv1, &tv);                                                   \
  time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  struct timeval tv1, tv2, tv;
  float time_delta;

  int *graph, *output_cpu, *output_gpu;
  int size;

  size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

  graph = (int *)malloc(size);
  assert(graph);

  output_cpu = (int *)malloc(size);
  assert(output_cpu);
  memset(output_cpu, 0, size);

  output_gpu = (int *)malloc(size);
  assert(output_gpu);

  generate_random_graph(graph, GRAPH_SIZE);

  fprintf(stderr, "running on cpu...\n");
  TIMER_START();
  floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  fprintf(stderr, "running on gpu...\n");
  TIMER_START();
  floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  if (memcmp(output_cpu, output_gpu, size) != 0) {
    fprintf(stderr, "FAIL!\n");
  }

  /* uncomment for debug
  printf("CPU - GPU\n");
  int i;
  for (i = 0; i < GRAPH_SIZE; i++) {
    printf("%2d - %2d\n", output_cpu[i], output_gpu[i]);
  }
  */

  return 0;
}
